#include "hip/hip_runtime.h"

#include <time.h>
#include <math.h>
#include <vector>
#include <memory>
#include <iostream>
#include <algorithm>
#define BIG (1e7)
// #define DEBUG

using namespace std;

template<typename T>
__global__ void mergeVec_half(T* A, T* tmp, const int64_t vSize) {
    int64_t left = blockIdx.x * vSize;
    int64_t right = left + vSize - 1;
    int64_t mid = (left + right) / 2;

    int64_t i = left, j = mid + 1, k = left;
    while ((i <= mid) && (j <= right)) {
        if (A[i] <= A[j]) {
            tmp[k++] = A[i++];
        } else {
            tmp[k++] = A[j++];
        }
    }
    if (i > mid) {
        for (; j <= right; ++j, ++k) {
            tmp[k] = A[j];
        }
    } else {
        for (; i <= mid; ++i, ++k) {
            tmp[k] = A[i];
        }
    }
    for (k = left; k <= right; ++k) {
        A[k] = tmp[k];
    }
}

template<typename theIterator, typename T>
void mergeSort_power2n(theIterator begin, theIterator end, T args) {
    clock_t begT, endT;

    T* dataA, * dataTmp;
    int64_t vSize = end - begin;
    hipMalloc((void**)&dataA, sizeof(*begin) * vSize);
    hipMalloc((void**)&dataTmp, sizeof(*begin) * vSize);

#ifdef DEBUG
    int64_t n = 0;
    if (vSize >= 2) {
        for (int64_t i = 1; i < vSize; i <<= 1) {
            n += 1;
        }
    } else {
        return;
    }
    if (((int64_t)1 << n) > vSize) {
        cerr << "\033[31;1m error! vSize != 2 ** n \033[0m";
        exit(-1);
    }
#endif

    begT = clock();
    hipMemcpy(dataA, &(*begin), sizeof(*begin) * vSize, hipMemcpyHostToDevice);

    for (int64_t i = 2; i <= vSize; i <<= 1) {
        mergeVec_half<<<vSize / i, 1>>>(dataA, dataTmp, i);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
    }

    hipMemcpy(&(*begin), dataA, sizeof(*begin) * vSize, hipMemcpyDeviceToHost);
    endT = clock();
    cout << "inside GPU operation, time = " << (endT - begT) / 
			(double)CLOCKS_PER_SEC << endl;

    hipFree(dataA);
    hipFree(dataTmp);
}

template<typename theIterator>
inline void mergeSort_power2n(theIterator begin, theIterator end) {
    mergeSort_power2n(begin, end, *begin);
}

template<typename theIterator, typename T>
void mergeVec(theIterator beg1, theIterator end1, theIterator beg2, theIterator end2, T args) {
    vector<T> tmp((end1 - beg1) + (end2 - beg2));
    theIterator i = beg1, j = beg2;
    auto k = tmp.begin();

    while (i != end1 && j != end2) {
        if (*i <= *j) {
            *k++ = *i++;
        } else {
            *k++ = *j++;
        }
    }
    while (i != end1) *k++ = *i++;
    while (j != end2) *k++ = *j++;

    k = tmp.begin();
    for (i = beg1; i != end1; ++i, ++k) {
        *i = *k;
    }
    for (j = beg2; j != end2; ++j, ++k) {
        *j = *k;
    }
}

template<typename theIterator>
inline void mergeVec(theIterator beg1, theIterator end1, theIterator beg2, theIterator end2) {
    mergeVec(beg1, end1, beg2, end2, *beg1);
}

template<typename vec>
void mergeSort_gpu(vec& A) {
    vector<bool> binA;
    int64_t vSize = A.size(), n = A.size();
    int64_t one = 1;
    while (n > 0) {
        binA.push_back(n & one);
        n >>= 1;
    }

    vector<int64_t> idxVec;
    idxVec.push_back(0);
    for (int64_t i = 0; i != binA.size(); ++i) {
        if (binA[i]) {
            idxVec.push_back(idxVec.back() + (one << i));
        }
    }

    for (int64_t i = 0; i != idxVec.size() - 1; ++i) {
        mergeSort_power2n(A.begin() + idxVec[i], A.begin() + idxVec[i + 1]);
    }

    for (int64_t i = 1; i != idxVec.size() - 1; ++i) {
        mergeVec(
            A.begin(), A.begin() + idxVec[i],
            A.begin() + idxVec[i], A.begin() + idxVec[i + 1]);
    }
}

template<typename theIterator, typename T>
void mergeSort_cpu(theIterator begin, theIterator end, T args) {
    if (end - begin <= 1) return;

    auto mid = begin + (end - begin) / 2;
    mergeSort_cpu(begin, mid, args);
    mergeSort_cpu(mid, end, args);

    vector<T> tmp(end - begin);
    auto i = begin, j = mid, k = tmp.begin();
    while (i != mid && j != end) {
        if (*i <= *j) {
            *k++ = *i++;
        } else {
            *k++ = *j++;
        }
    }
    while (i != mid) *k++ = *i++;
    while (j != end) *k++ = *j++;

    k = tmp.begin();
    for (i = begin; i != end; ++i, ++k) {
        *i = *k;
    }
}

template<typename theIterator>
inline void mergeSort_cpu(theIterator begin, theIterator end) {
    mergeSort_cpu(begin, end, *begin);
}

int main(int argc, char* argv[])
{
	if (argc < 2) {
        cout << "Usage: ./<program> <number of n>" << endl;
        return 1;
    }
    int n = atoi(argv[1]);

    vector<int> a(n), b(n), c(n);
    for (int i = 0; i < n; ++i) {
        cin >> a[i];
    }
    b = a;
    c = a;
    
    clock_t t1, t2;
	t1 = clock();
    mergeSort_gpu(a);
    t2 = clock();

	cout << "CUDA Time taken: " << (t2 - t1) / 
			(double)CLOCKS_PER_SEC << endl;

	t1 = clock();
    mergeSort_cpu(b.begin(), b.end());
    t2 = clock();

	cout << "CPU Time taken: " << (t2 - t1) / 
			(double)CLOCKS_PER_SEC << endl;

	t1 = clock();
    stable_sort(c.begin(), c.end());
    t2 = clock();

	cout << "STL Time taken: " << (t2 - t1) / 
			(double)CLOCKS_PER_SEC << endl;
}